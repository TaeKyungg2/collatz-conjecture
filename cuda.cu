#include "hip/hip_runtime.h"
#include <cstdlib>
#include <fstream>
#include <cstring>
using namespace std;

__device__ void cal(unsigned long long int n,unsigned long long int road[1000],unsigned int x);
__device__ __managed__ unsigned long long int all_road[10000][1000]={};


__global__ void go_cal(){
    for(unsigned int i=2;i<=10000;i++){
        unsigned long long int road[1000]={};
        unsigned int x=0;
        cal(i,road,x);
    }
}
int main(){
    fstream f;
    f.open("../col_dataset_100m.txt", ios::in | ios::out);
    hipDeviceSetLimit(hipLimitStackSize, 256 * 1024);
    go_cal<<<1024,256>>>();
    hipDeviceSynchronize();
    hipMemcpy(host_array, device_array, x * sizeof(unsigned long long), hipMemcpyDeviceToHost);
    for(int i=2;i<10000;i++){
        for(int j=1;j<1000;j++){
            if(all_road[i][j]!=0){
                f << all_road[i][j] << ",";
            }
            else{break;}
        }
        f << endl;
    }
    f.close();
    return 0;
}

__device__ unsigned long long int big_int=18446744073709551614;

__device__ void cal(unsigned long long int n,unsigned long long int road[1000],unsigned int x){
    x+=1;
    if(x>=1000){printf("%llu over 1000 count.",n);}
    road[x]=n;
    if(n==1){
        for(int i=1;i<1000;i++){
            if(road[i]==0){ return;}
            all_road[road[1]][i]=road[i];
        }
        return;
    }
    if(n%2==0){
        cal(n>>1,road,x);
    }
    else{
        if(n>=big_int){
            printf("start_num: %llu over",road[1]);
            return;
        }
        cal((n<<1)+n+1,road,x);
    }
}